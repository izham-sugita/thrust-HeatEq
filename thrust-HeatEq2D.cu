#include "hip/hip_runtime.h"
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>

#include<iostream>
#include<iomanip>
#include<cstdlib>
#include<cstdio>
#include<cmath>
#include<fstream>
#include<vector>
#include<chrono>

#include<blocksize.h> //contains blockDimX, blockDimY

#define pi 4.0*atan(1.0)

using namespace std;
using namespace std::chrono;

void   swap
// ====================================================================
//
// purpos     :  update the variable fn --> f
//
// date       :  Jul 03, 2001
// programmer :  Takayuki Aoki
// place      :  Tokyo Institute of Technology
//
(
   float   **f,        /* dependent variable                        */
   float   **fn        /* updated variable                          */
)
// --------------------------------------------------------------------
{
     float  *tmp = *f;   *f = *fn;   *fn = tmp;
}


//Heat equation kernel
__global__  void  HeatEq2D
// ====================================================================
//
// program    :  Two-dimensional heat equation kernel
//
// date       :  2018/09/15
// programmer :  Muhammad Izham 
// place      :  Universiti Malaysia Perlis
//
(
   float    *f,         /* dependent variable                        */
   float    *fn,        /* dependent variable                        */
   int      nx,         /* grid number in the x-direction            */
   int      ny,         /* grid number in the x-direction            */
   float    c0,         /* coefficient no.0                          */
   float    c1,         /* coefficient no.1                          */
   float    c2          /* coefficient no.2                          */
)
// --------------------------------------------------------------------
{
   int    j,    jx,   jy;
   float  fcc,  fce,  fcw,  fcs,  fcn;

   jy = blockDim.y*blockIdx.y + threadIdx.y;
   jx = blockDim.x*blockIdx.x + threadIdx.x;
   /*Dirichilet BC, fixed boundary*/
if(jx > 0  && jx < nx-1){
if(jy > 0 && jy < ny-1){
  j = nx*jy + jx;
     fcc = f[j];
     fcw = f[j - 1];
     fce = f[j+1];
     fcs = f[j-nx];
     fcn = f[j+nx];
     
   fn[j] = c0*(fce + fcw)
         + c1*(fcn + fcs)
         + c2*fcc;
 }
}

}

int main()
{

  int imax = 128;
  int jmax = 128;

  cout<<"Enter imax, jmax \n";
  cin>>imax>>jmax;
  
  float dx = 1.0f /(float)(imax-1);
  float dy = 1.0f /(float)(jmax-1);
  float dt = 0.01f*dx*dx;

  //test std::vector
  vector<float> h_test;
  h_test.resize(imax*jmax);
    for(int i=0; i<imax; ++i){
    for(int j=0; j<jmax; ++j){
      int id = i*jmax + j;
      h_test[id] = sin((float)i*pi*dx )*sin((float)j*pi*dy );
    }
  }
  
  thrust::host_vector<float> h_Told(imax*jmax);
  thrust::host_vector<float> h_Tnew(imax*jmax); 

  /*Initiate host vector*/
  for(int i=0; i<imax; ++i){
    for(int j=0; j<jmax; ++j){
      int id = i*jmax + j;
      h_Told[id] = sin((float)i*pi*dx )*sin((float)j*pi*dy );
      h_Tnew[id] = 0.0f;
    }
  }

  ofstream finit;
  finit.open("initHeat2D.csv");
  finit << "x, y, z, Temp\n";
  finit << setprecision(8);
  finit << fixed;
  for(int i=0; i<imax; ++i){
    for(int j=0; j<jmax; ++j){
      int id = i*jmax + j;
      finit<<(float)i*dx<<","
	<<(float)j*dy<<","
	<<h_test[id]<<","
	<<h_test[id]<<endl;
    }
  }
  finit.close();

  FILE *fp0;
  fp0 = fopen("initHeatOld.csv","w");
  fprintf(fp0,"x, y, z, temp\n");
  for(int i=0; i<imax; ++i){
    for(int j=0; j<jmax; ++j){
      int id = i*jmax + j;
      float xg = (float)i*dx;
      float yg = (float)j*dy;
      fprintf(fp0,"%f, %f, %f, %f\n", xg, yg, h_test[id], h_test[id]);
	}
  }
  fclose(fp0);
  
  thrust::device_vector<float> d_Told = h_Told;
  thrust::device_vector<float> d_Tnew = h_Tnew;
  
  float kappa = 1.0f;
  float c0 = kappa*dt/(dx*dx), c1 = kappa*dt/(dy*dy),
               c2 = 1.0 - 2.0*(c0 + c1);

  float *d_ToldPointer = thrust::raw_pointer_cast(&d_Told[0]);
  float *d_TnewPointer = thrust::raw_pointer_cast(&d_Tnew[0]);

  dim3 grid(imax/blockDimX, jmax/blockDimY, 1);
  dim3 threads(blockDimX, blockDimY, 1);
  
  int itermax =20000;
  double flops = 0.0f;

  high_resolution_clock::time_point t1=high_resolution_clock::now();  
  
  for(int iter=0; iter<itermax; ++iter){
  /*Calling kernel*/
  HeatEq2D<<<grid,threads>>>(d_ToldPointer,
				       d_TnewPointer,
				       imax,jmax,c0,c1,c2);

  
  //Update device_vector
  //d_Told = d_Tnew;

  swap(&d_ToldPointer,&d_TnewPointer); //the best way to swap!
  
  flops = flops + 7.0*((float)imax * (float)jmax);
  
  }/*end time loop*/

  high_resolution_clock::time_point t2=high_resolution_clock::now();

  duration<double> elapsed_time = duration_cast< duration<double> >(t2-t1);

  double timing = elapsed_time.count();

  cout<<"Total operations: "<<flops<<endl;

  flops = flops/(timing*1.0e9);
  cout<<"Elapsed time for "<<itermax<<" steps is "<<timing<<" secs."<<endl;
  cout<<"Performance: "<<flops<<" GFLOPS"<<endl;
  
  /*copy data back to host*/
  h_Tnew = d_Told;
  
  /*output data .csv*/
  ofstream fp;
  fp.open("thrustHeat.csv");
  fp << "x, y, z, Temp\n";
  fp << setprecision(8);
  fp << fixed;
  for(int i=0; i<imax; ++i){
    for(int j=0; j<jmax; ++j){
      int id = i*jmax + j;
      fp<<(float)i*dx<<","
	<<(float)j*dy<<","
	<<h_Tnew[id]<<","
	<<h_Tnew[id]<<endl;
    }
  }
  fp.close();
  
}
